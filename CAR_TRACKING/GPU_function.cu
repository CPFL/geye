#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "for_use_GPU.h"
#include "calc_feature_conf.h"


/* declaration of texture memory */
//texture<FLOAT> A;
//texture<FLOAT> B;
texture<float, hipTextureType1D, hipReadModeElementType> A;
texture<float, hipTextureType1D, hipReadModeElementType> B;
texture<int2, hipTextureType1D, hipReadModeElementType> A_double;
texture<int2, hipTextureType1D, hipReadModeElementType> B_double;


//thread process
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// convolve A and B(non_symmetric)
//unsigned __stdcall process(void *thread_arg) {

/********************************************/
/* function for calculating root */
/********************************************/
extern "C"
__global__
void
process_root 
(
 //FLOAT *A,  
 //FLOAT *B, 
 FLOAT *C, 
 int *A_dims_array, 
 int *B_dims_array, 
 int len,
 int interval, 
 int L_MAX,
 int *error_array,
 int error_array_num,
 int pid,
 int device_number
) 
{
  int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  int ii = blockIdx.z % len;
  int level = blockIdx.z / len;

  int A_dims[3] = { A_dims_array[level*3], A_dims_array[level*3+1], A_dims_array[level*3+2] };
  int B_dims[3] = { B_dims_array[ii*3], B_dims_array[ii*3+1], B_dims_array[ii*3+2] };
  int C_dims[2] = { A_dims[0] - B_dims[0] + 1, A_dims[1] - B_dims[1] + 1 };

  int C_x = C_dims[1]/device_number;
  
  if(C_dims[1]%device_number != 0){
    C_x++;
  }
 
  idx_x = idx_x + pid * C_x;
 
  if(idx_x < C_x * pid  ||  idx_x >=  C_x * (pid + 1)){
    return ;
  }  

  if(0 <= ii && ii < len && 0 <= idx_x && idx_x < C_dims[1] && 0 <= idx_y && idx_y < C_dims[0] && interval <= level && level < L_MAX ) { 


    int num_features = A_dims[2];
    const int A_SQ = A_dims[0]*A_dims[1];
    const int B_SQ = B_dims[0]*B_dims[1];
    FLOAT add_val = 0;
    
    int x = idx_x;
    int y = idx_y;
    int XA0 = A_dims[0]*x;

    
    /* apply loop condition */
    for(int i=0; i<error_array_num; i++){
      if(error_array[i] == level){
        return;
      }
    }
    
    
    
    /* adjust the location of pointer of C */
    FLOAT *dst;
    unsigned long long int pointer = (unsigned long long int)C;

    for(int a=interval; a<level; a++) {
      for(int b=0; b<len; b++) {
        int height = A_dims_array[a*3] - B_dims_array[b*3] + 1; 
        int width = A_dims_array[a*3 + 1] - B_dims_array[b*3 + 1] + 1;
        
        /* error semantics */
        if (height < 1 || width < 1){
          printf("Invalid input in GPU\n");
          return;
        }
        
        pointer += (unsigned long long int)(height*width*sizeof(FLOAT));
       
      }
    }

    for(int b=0; b<ii; b++){
      int height = A_dims_array[level*3] - B_dims_array[b*3] + 1;
      int width  = A_dims_array[level*3 + 1] - B_dims_array[b*3 + 1] + 1;

      /* error semantics */
      if (height < 1 || width < 1){
        printf("Invalid input in GPU\n");
        return;
      }
      
      pointer += (unsigned long long int)(height*width*sizeof(FLOAT));
    }
    
    dst = (FLOAT *)pointer;
    
    /* adjust the location of pointer of A */
    //unsigned long long int pointerA = (unsigned long long int)A;
    int A_index_ini = 0;
    for(int a=0; a<level; a++) {
      //      pointerA += (unsigned long long int)(A_dims_array[a*3]*A_dims_array[a*3 + 1]*A_dims_array[a*3 + 2]*sizeof(FLOAT));
      A_index_ini += A_dims_array[a*3]*A_dims_array[a*3 + 1]*A_dims_array[a*3 + 2];
    }
    
    
    /* adjust the location of pointer of B */
    //unsigned long long int pointerB = (unsigned long long int)B;
    int B_index_ini = 0;
    for(int b=0; b<ii; b++) {
      //      pointerB += (unsigned long long int)(B_dims_array[b*3]*B_dims_array[b*3 + 1]*B_dims_array[b*3 + 2]*sizeof(FLOAT));
      B_index_ini += B_dims_array[b*3]*B_dims_array[b*3 + 1]*B_dims_array[b*3 + 2];
    } 

            
    for(int f = 0; f < num_features; f++) // num_features = 31
      {  
        // FLOAT *A_src = (FLOAT *)pointerA + f*A_SQ;      
        int A_index = A_index_ini + f*A_SQ;
        // FLOAT *B_src = (FLOAT *)pointerB + f*B_SQ;     
        int B_index = B_index_ini + f*B_SQ;
        
        // FLOAT *A_src2 =A_src+XA0; 
        A_index += XA0;

        FLOAT val = 0;
        // FLOAT *A_off = A_src2+y;
        A_index += y;
        // FLOAT *B_off = B_src;
        
        for (int xp = 0; xp < B_dims[1]; xp++) 
          {
            // FLOAT *A_temp = A_off;						
            int A_index_tmp = A_index;
            // FLOAT *B_temp = B_off;
            int B_index_tmp = B_index;
	  
            for (int yp = 0; yp < B_dims[0]; yp++) 	  
              {
                // val += *(A_temp++) * *(B_temp++);
                if(sizeof(FLOAT) == sizeof(float)) // if configured to use single precision
                  {
                    FLOAT A_val = tex1Dfetch(A, A_index_tmp);
                    FLOAT B_val = tex1Dfetch(B, B_index_tmp);
                    val += A_val * B_val;
                  } 
                else
                  {      // if configured to use double precision
                    int2 A_val = tex1Dfetch(A_double, A_index_tmp);
                    int2 B_val = tex1Dfetch(B_double, B_index_tmp);
                    val += __hiloint2double(A_val.y, A_val.x) * __hiloint2double(B_val.y, B_val.x);
                  }
                
                A_index_tmp++;
                B_index_tmp++;
              }
            
            // A_off+=A_dims[0];
            A_index += A_dims[0];
            // B_off+=B_dims[0];
            B_index += B_dims[0];
            
          }
        
        add_val += val;
      }
    
    *(dst + (idx_x*C_dims[0] + idx_y)) += add_val;
  }
  
  
  return;
}



/********************************************/
/* function for calculating part */
/********************************************/
extern "C"
__global__
void
process_part
(
 //FLOAT *A,  
 //FLOAT *B, 
 FLOAT *C, 
 int *A_dims_array, 
 int *B_dims_array, 
 int len,
 int interval, 
 int L_MAX,
 int *error_array,
 int error_array_num,
 int pid,
 int device_number
) 
{
  int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  int ii = blockIdx.z % len;
  int level = blockIdx.z / len; 

  int A_dims[3] = { A_dims_array[level*3], A_dims_array[level*3+1], A_dims_array[level*3+2] };
  int B_dims[3] = { B_dims_array[ii*3], B_dims_array[ii*3+1], B_dims_array[ii*3+2] };
  int C_dims[2] = { A_dims[0] - B_dims[0] + 1, A_dims[1] - B_dims[1] + 1 };

  int C_x = C_dims[1]/device_number;

  if(C_dims[1]%device_number != 0){
    C_x++;
  }  
 
  idx_x = idx_x + pid * C_x;
 
  if(idx_x < C_x * pid  ||  idx_x >=  C_x * (pid + 1)){
    return ;
  }  

  if(0 <= ii && ii < len && 0 <= idx_x && idx_x < C_dims[1] && 0 <= idx_y && idx_y < C_dims[0] && 0 <= level && level < (L_MAX - interval) ) {
    int num_features = A_dims[2];
    const int A_SQ = A_dims[0]*A_dims[1];
    const int B_SQ = B_dims[0]*B_dims[1];
    FLOAT add_val = 0;

    int x = idx_x;
    int y = idx_y;
    int XA0 = A_dims[0]*x;
    
    /* apply loop condition */
    for(int i=0; i<error_array_num; i++){
      if(error_array[i] == level)
        return;
    }
    
    /* adjust the location of pointer of C */
    FLOAT *dst;
    unsigned long long int pointer = (unsigned long long int)C;
    for(int a=0; a<level; a++) {
      for(int b=0; b<len; b++){
        int height = A_dims_array[a*3] - B_dims_array[b*3] + 1;
        int width = A_dims_array[a*3 + 1] - B_dims_array[b*3 + 1] + 1;
        
        /* error semantics */
        if(height < 1 || width < 1){
          printf("Invalid input in GPU\n");
          return;
        }
        
        pointer += (unsigned long long int)(height*width*sizeof(FLOAT));
      }
    }

    for(int b=0; b<ii; b++){
      int height = A_dims_array[level*3] - B_dims_array[b*3] + 1;
      int width  = A_dims_array[level*3 + 1] - B_dims_array[b*3 + 1] + 1;

       /* error semantics */
        if(height < 1 || width < 1){
          printf("Invalid input in GPU\n");
          return;
        }

      pointer += (unsigned long long int)(height*width*sizeof(FLOAT));
    }
    

    dst = (FLOAT *)pointer;

    /* adjust the location of pointer of A */
    // unsigned long long int pointerA = (unsigned long long int)A;
    int A_index_ini = 0;
    for(int a=0; a<level; a++) {
      // pointerA += (unsigned long long int)(A_dims_array[a*3]*A_dims_array[a*3 + 1]*A_dims_array[a*3 + 2]*sizeof(FLOAT));
      A_index_ini += A_dims_array[a*3]*A_dims_array[a*3 + 1]*A_dims_array[a*3 + 2];
    }
    
    /* adjust the location of pointer of B */
    // unsigned long long int pointerB = (unsigned long long int)B;
    int B_index_ini = 0;
    for(int b=0; b<ii; b++) {
      // pointerB += (unsigned long long int)(B_dims_array[b*3]*B_dims_array[b*3 + 1]*B_dims_array[b*3 + 2]*sizeof(FLOAT));
      B_index_ini += B_dims_array[b*3]*B_dims_array[b*3 + 1]*B_dims_array[b*3 + 2];
    } 
    
    for(int f = 0; f < num_features; f++) // num_features = 31
      {  
        // FLOAT *A_src = (FLOAT *)pointerA + f*A_SQ;      
        int A_index = A_index_ini + f*A_SQ;
        // FLOAT *B_src = (FLOAT *)pointerB + f*B_SQ;     
        int B_index = B_index_ini + f*B_SQ;
        
        // FLOAT *A_src2 =A_src+XA0; 
        A_index += XA0;

        FLOAT val = 0;
        // FLOAT *A_off = A_src2+y;
        A_index += y;
        // FLOAT *B_off = B_src;
        
        for (int xp = 0; xp < B_dims[1]; xp++) 
          {
            // FLOAT *A_temp = A_off;						
            int A_index_tmp = A_index;
            // FLOAT *B_temp = B_off;	  
            int B_index_tmp = B_index;
 
            for (int yp = 0; yp < B_dims[0]; yp++) 	  
              {
                // val += *(A_temp++) * *(B_temp++);
                if(sizeof(FLOAT) == sizeof(float)) // if configured to use single precision
                  {
                    FLOAT A_val = tex1Dfetch(A, A_index_tmp);
                    FLOAT B_val = tex1Dfetch(B, B_index_tmp);
                    val += A_val * B_val;
                  }
                else            // if configured to use double precision
                  {
                    int2 A_val = tex1Dfetch(A_double, A_index_tmp);
                    int2 B_val = tex1Dfetch(B_double, B_index_tmp);
                    val += __hiloint2double(A_val.y, A_val.x) * __hiloint2double(B_val.y, B_val.x);
                  }
                
                A_index_tmp++;
                B_index_tmp++;
              }
            
            // A_off+=A_dims[0];
            A_index += A_dims[0];
            // B_off+=B_dims[0];
            B_index += B_dims[0];
            
          }
        add_val += val;
      }

    *(dst + (idx_x*C_dims[0] + idx_y)) += add_val;
  }
  
  return;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__
void
inverse_Q(
  FLOAT *src_start,
  int *size_array,
  int *error_array,
  int error_array_num,
  int NoP,
  int *PIDX_array,
  int *numpart,
  int NoC,
  int max_numpart,
  int interval,
  int L_MAX,
  int pid,
  int device_number
          )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int kk = blockIdx.y * blockDim.y + threadIdx.y;
  int jj = threadIdx.z;
  int L = blockIdx.z;
  int numpart_jj;
  int C_y;


  if(0<=jj && jj<NoC)
    {
      numpart_jj = numpart[jj];
      C_y = numpart_jj/device_number;
      if(numpart_jj%device_number != 0){
        C_y++;
       }
      kk = kk + pid * C_y;
      if(kk < C_y * pid  ||  kk >=  C_y * (pid + 1)){
         return ;
       }
    } else return ;
   

  if(0<=L && L < (L_MAX-interval)) 
    {
  
      /* loop condition */
      for(int h=0; h<error_array_num; h++) {
        if(L==error_array[h]){ 
          return;
        }
      }
    
     
      if( 0<=kk && kk < numpart_jj )
        {
          int PIDX = PIDX_array[L*(NoC*max_numpart) + jj*max_numpart + kk];
          int dim0 = size_array[L*NoP*2 + PIDX*2];
          int dim1 = size_array[L*NoP*2 + PIDX*2+1]; 

          if( idx < 0 || dim0*dim1 <= idx) return;

              /* pointer adjustment */
          FLOAT *src;
          unsigned long long int ptr_adjuster = (unsigned long long int)src_start;
          for(int i=0; i<L; i++) {
                
                /* apply error condition */
            int error_flag=0;
            for(int h=0; h<error_array_num; h++) {
              if(i==error_array[h]){
                error_flag = 1;
              }
            }
            if(error_flag != 0) {
              continue;
            }

                
            for(int j=0; j<NoP; j++) {
              int height = size_array[i*NoP*2 + j*2];
              int width = size_array[i*NoP*2 + j*2+1];
              ptr_adjuster += (unsigned long long int)(height*width*sizeof(FLOAT));
                  
            }
          }
              
   
              
          for(int j=0; j<PIDX; j++) {
            int height = size_array[L*NoP*2 + j*2];
            int width = size_array[L*NoP*2 + j*2+1];
            ptr_adjuster += (unsigned long long int)(height*width*sizeof(FLOAT));
          }
              
          src = (FLOAT *)ptr_adjuster;  
                        
          *(src + idx) *= -1;
        
      }
    }       
}



///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// dt helper function
__device__
void 
dt_helper(FLOAT *src, FLOAT *dst, int *ptr, int step, int s1, int s2, int d1, int d2, FLOAT a, FLOAT b) 
{
  if (d2 >= d1) 
    {
      int d = (d1+d2) >> 1;
      int ds =d*step;
      int s = s1;
      FLOAT src_ss = *(src+s*step);
      for (int p = s1+1; p <= s2; p++)
        {
          int t1 = d-s;
          int t2 = d-p;
          if (src_ss + a*t1*t1 + b*t1 > *(src+p*step) + a*t2*t2 + b*t2) 
            {
              s = p;
              src_ss = *(src+s*step);
            }
        }
      int D = d-s;
      dst[ds] = *(src+s*step) + a*D*D + b*D;
      ptr[ds] = s;
      dt_helper(src, dst, ptr, step, s1, s, d1, d-1, a, b);
      dt_helper(src, dst, ptr, step, s, s2, d+1, d2, a, b);
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//sub function of dt 
extern "C"
__global__
void 
dt1d_x(
  FLOAT *src_start,             // part_C_dev
  FLOAT *dst_start,             // tmpM_dev
  int *ptr_start,               // tmpIy_dev
  int *DID_4_array,             // DID_4_array_dev
  FLOAT *def_array,             // def_array_dev
  int *size_array,              // pm_size_array_dev
  int NoP,                      // NoP
  int *PIDX_array,              // PIDX_array_dev
  int *error_array,             // part_error_array_dev
  int error_array_num,          // part_error_array_num
  int *numpart,                 // numpart_jj
  int NoC,                      // NoC
  int max_numpart,              // max_numpart
  int interval,                 // interval
  int L_MAX,                     // L_MAX
  int pid,                       // pid
  int device_number              // device_number

       ) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int kk = blockIdx.y * blockDim.y + threadIdx.y;
  int jj = threadIdx.z;
  int L = blockIdx.z;
  int numpart_jj;
  int C_y;

  if(0<=jj && jj<NoC)
    {

      numpart_jj = numpart[jj];
      C_y = numpart_jj/device_number;

      if(numpart_jj%device_number != 0){
        C_y++;
       }
 
      kk = kk + pid * C_y;
 
      if(kk < C_y * pid  ||  kk >=  C_y * (pid + 1)){
         return ;
       }
    } else{
      return ;
    }


  if(0<=L && L<(L_MAX-interval)) 
    {
      /* loop condition */
      for(int h=0; h<error_array_num; h++) {
        if(L==error_array[h]){ 
          return;
        }
      }
                
      if(0<=kk && kk<numpart_jj)
        {
          int PIDX = PIDX_array[L*(NoC*max_numpart) + jj*max_numpart + kk];
          int dim1 = size_array[L*NoP*2 + PIDX*2+1]; 

          if( idx < 0 || dim1 <= idx ) return;

          int dim0 = size_array[L*NoP*2 + PIDX*2];
          int XD=0;
          int step = 1;
          int n = dim0;  
          int DID_4 = DID_4_array[L*(NoC*max_numpart) + jj*max_numpart + kk];
          FLOAT a = def_array[DID_4+2];
          FLOAT b = def_array[DID_4+3];
             
          /* pointer adjustment */
          unsigned long long int adj_src = (unsigned long long int)src_start;
          unsigned long long int adj_dst = (unsigned long long int)dst_start;
          unsigned long long int adj_ptr = (unsigned long long int)ptr_start;
          /* for src */
          for(int i=0; i<L; i++) {
                
            /* apply error condition */
            int error_flag=0;
            for(int h=0; h<error_array_num; h++) {
              if(i==error_array[h]){
                error_flag = 1;
              }
            }
            if(error_flag != 0) {
              continue;
            }
                
            for(int j=0; j<NoP; j++) {
              int height = size_array[i*NoP*2 + j*2];
              int width = size_array[i*NoP*2 + j*2+1];
              adj_src += (unsigned long long int)(height*width*sizeof(FLOAT));
                  
            }
          }
              
              
          for(int j=0; j<PIDX; j++) {
            int height = size_array[L*NoP*2 + j*2];
            int width = size_array[L*NoP*2 + j*2+1];
            adj_src += (unsigned long long int)(height*width*sizeof(FLOAT));
          }
              
              /* for dst, ptr */
              // adjust "dst" to tmpM[L][jj][kk]
              // adjust "ptr" to tmpIy[L][jj][kk]
          for(int i=0; i<L; i++) {
                
                /* apply error condition */
            int error_flag=0;
            for(int h=0; h<error_array_num; h++) {
              if(i==error_array[h]){
                error_flag = 1;
              }
            }
            if(error_flag != 0) {
              continue;
            }
                
            for(int j=0; j<NoC; j++) {
              for(int k=0; k<numpart[j]; k++) {
                int PIDX_tmp = PIDX_array[i*(NoC*max_numpart) + j*max_numpart + k];
                int dims0_tmp = size_array[i*NoP*2 + PIDX_tmp*2];
                int dims1_tmp = size_array[i*NoP*2 + PIDX_tmp*2+1];

                    
                adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
                adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
                    
                    
              }
            }
          }
              

          for(int i=0; i<jj; i++) {
            for(int j=0; j<numpart[i]; j++) {
              int PIDX_tmp = PIDX_array[L*(NoC*max_numpart) + i*max_numpart + j]; // PIDX_array[L][i][j]
              int dims0_tmp = size_array[L*NoP*2 + PIDX_tmp*2]; // size_array[L][PIDX_tmp*2]
              int dims1_tmp = size_array[L*NoP*2 + PIDX_tmp*2+1]; // size_array[L][PIDX_tmp*2+1]
                  
              adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
              adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
                  
            }
          }
              
          for(int j=0; j<kk; j++) {
            int PIDX_tmp = PIDX_array[L*(NoC*max_numpart) + jj*max_numpart + j]; // PIDX_array[L][jj][j]
            int dims0_tmp = size_array[L*NoP*2 + PIDX_tmp*2]; // size_array[L][PIDX_tmp*2]
            int dims1_tmp = size_array[L*NoP*2 + PIDX_tmp*2+1]; // size_array[L][PIDX_tmp*2+1]
                
            adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
            adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
          }
              
              
          FLOAT *src = (FLOAT *)adj_src;
          FLOAT *dst = (FLOAT *)adj_dst;
          int *ptr = (int *)adj_ptr;
              
          /* main calculation of di1d_x */
          XD = idx*dim0;
          dt_helper(src+XD, dst+XD, ptr+XD, step, 0, n-1, 0, n-1, a, b);
            
        }
    }
}


extern "C"
__global__
void 
dt1d_y(
  FLOAT *src_start,             // tmpM_dev
  FLOAT *dst_start,             // M_dev
  int *ptr_start,               // tmpIx_dev
  int *DID_4_array,             // DID_4_array_dev
  FLOAT *def_array,             // def_array_dev
  int NoP,                      // NoP
  int *size_array,              // pm_size_array_dev
  int *numpart,                 // numpart_jj
  int *PIDX_array,              // PIDX_array_dev
  int NoC,                      // NoC
  int max_numpart,              // max_numpart
  int interval,                 // interval
  int L_MAX,                    // L_MAX
  int *error_array,             // part_error_array_dev
  int error_array_num,           // part_error_array_num
  int pid,                       // pid
  int device_number              // device_number
       ) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int kk = blockIdx.y * blockDim.y + threadIdx.y;
  int jj = threadIdx.z;
  int L = blockIdx.z;
  int numpart_jj;
  int C_y;

  if(0<=jj && jj<NoC)
    {

      numpart_jj = numpart[jj];
      C_y = numpart_jj/device_number;

      if(numpart_jj%device_number != 0){
        C_y++;
       }
 
      kk = kk + pid * C_y;
 
      if(kk < C_y * pid  ||  kk >=  C_y * (pid + 1)){
         return ;
       }
    } else{
      return ;
    }


  if(0<=L && L<(L_MAX-interval)) 
    {
      /* loop condition */
      for(int h=0; h<error_array_num; h++) {
        if(L==error_array[h]){ 
          return;
        }
      }
      
      
      if( 0<=kk && kk<numpart_jj)
        {
          int PIDX = PIDX_array[L*(NoC*max_numpart) + jj*max_numpart + kk];
          int dim0 = size_array[L*NoP*2 + PIDX*2];

          if( idx < 0 || dim0 <= idx ) return;

          int dim1 = size_array[L*NoP*2 + PIDX*2+1];
          int step  = dim0;
          int n = dim1;
              
          int DID_4 = DID_4_array[L*(NoC*max_numpart) + jj*max_numpart + kk];
              
          FLOAT a = def_array[DID_4];   // ax
          FLOAT b = def_array[DID_4+1]; // bx
              
              /* pointer adjustment */
          unsigned long long int adj_src = (unsigned long long int)src_start;
          unsigned long long int adj_dst = (unsigned long long int)dst_start;
          unsigned long long int adj_ptr = (unsigned long long int)ptr_start;
              /* for src, dst, ptr */
              /* adjust "src" to tmpM[L][jj][kk] */
              /* adjust "dst" to M[L][jj][kk] */
              /* adjust "ptr" to tmpIx[L][jj][kk] */
          for(int i=0; i<L; i++) {

            /* apply error condition */
            int error_flag=0;
            for(int h=0; h<error_array_num; h++) {
              if(i==error_array[h]){
                error_flag = 1;
              }
            }
            if(error_flag != 0) {
              continue;
            }
                
            for(int j=0; j<NoC; j++) {
              for(int k=0; k<numpart[j]; k++) {
                    
                int PIDX_tmp = PIDX_array[i*(NoC*max_numpart) + j*max_numpart + k];
                int dims0_tmp = size_array[i*NoP*2 + PIDX_tmp*2];
                int dims1_tmp = size_array[i*NoP*2 + PIDX_tmp*2+1];
                    
                adj_src += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
                adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
                adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
                    
              }
            }
          }


          for(int i=0; i<jj; i++) {
            for(int j=0; j<numpart[i]; j++) {
              int PIDX_tmp = PIDX_array[L*(NoC*max_numpart) + i*max_numpart + j]; // PIDX_array[L][i][j]
              int dims0_tmp = size_array[L*NoP*2 + PIDX_tmp*2]; // size_array[L][PIDX_tmp*2]
              int dims1_tmp = size_array[L*NoP*2 + PIDX_tmp*2+1]; // size_array[L][PIDX_tmp*2+1]
                  
              adj_src += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
              adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
              adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
                  
            }
          }
              
          for(int j=0; j<kk; j++) {
            int PIDX_tmp = PIDX_array[L*(NoC*max_numpart) + jj*max_numpart + j];
            int dims0_tmp = size_array[L*NoP*2 + PIDX_tmp*2];
            int dims1_tmp = size_array[L*NoP*2 + PIDX_tmp*2+1];
                
            adj_src += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
            adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
            adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
          }
              
              
              
          FLOAT *src = (FLOAT *)adj_src;
          FLOAT *dst = (FLOAT *)adj_dst;
          int *ptr = (int *)adj_ptr;
              
 
          dt_helper(src+idx, dst+idx, ptr+idx, step, 0, n-1, 0, n-1, a, b);
          
              
        }
    }
}

/*************************************************************/
/*************************************************************/
/* original source of dt function loop */
// for (int x = 0; x < dims[1]; x++)
//   {
//     dt1d(vals+XD, tmpM+XD, tmpIy+XD, 1, dims[0], ay, by);
//     XD+=dims[0];
//   }
// for (int y = 0; y < dims[0]; y++)
//   {
//     dt1d(tmpM+y, M+y, tmpIx+y, dims[0], dims[1], ax, bx);
//   }
/*************************************************************/
/*************************************************************/



extern "C"
__global__
void
calc_a_score(
 int IWID,
 int IHEI,
 FLOAT scale,
 int padx_n,
 int pady_n,
 int *RX_array,
 int *RY_array,
 FLOAT *ac_score,
 FLOAT *score_array,
 int *ssize_array,
 int NoC,
 int *size_score_array
)
{
  int ii = blockIdx.x * blockDim.x + threadIdx.x;
  int jj = blockIdx.y * blockDim.y + threadIdx.y;

  int component_jj = threadIdx.z;

  if(0<=component_jj && component_jj < NoC) 
    {

      unsigned long long int pointer_score = (unsigned long long int)score_array;
      unsigned long long int pointer_ssize = (unsigned long long int)ssize_array;
      unsigned long long int pointer_RX = (unsigned long long int)RX_array;
      unsigned long long int pointer_RY = (unsigned long long int)RY_array;
      for(int k=0; k<component_jj; k++) {
        pointer_score += (unsigned long long int)size_score_array[k];
        pointer_ssize += (unsigned long long int)(sizeof(int));
        pointer_RX += (unsigned long long int)(sizeof(int));
        pointer_RY += (unsigned long long int)(sizeof(int));
      }

      FLOAT *score = (FLOAT *)pointer_score;
      int ssize0 = *((int *)pointer_ssize);
      int ssize1 = *((int *)pointer_ssize + sizeof(int));
      int RX = *((int *)pointer_RX);
      int RY = *((int *)pointer_RY);



      if(0<=ii && ii<IWID && 0<=jj && jj<IHEI)
        {
          int Xn = (int)((FLOAT)ii/scale+padx_n);
          int Yn = (int)((FLOAT)jj/scale+pady_n);

          
          if(Yn<ssize0 && Xn<ssize1)
            {
              FLOAT sc = score[Yn+Xn*ssize0];
              int Im_Y = jj+RY;
              int Im_X = ii+RX;
              if(Im_Y<IHEI && Im_X<IWID)
                {
                  FLOAT *PP = ac_score+Im_Y+Im_X*IHEI;
                  if(sc>*PP) *PP=sc;
                }
            }
        }
    }
  
  /*************************************************************/
  /*************************************************************/
  /* original source of calc_a_score loop */
  // for(int ii=0;ii<IWID;ii++)
  //   {
  //     int Xn=(int)((FLOAT)ii/scale+padx_n);
  
  //     for(int jj=0;jj<IHEI;jj++)
  //       {
  //         int Yn =(int)((FLOAT)jj/scale+pady_n);
  
  //         if(Yn<ssize[0] && Xn<ssize[1])
  //           {
  //             FLOAT sc = score[Yn+Xn*ssize[0]]; //get score of pixel
      
  //             int Im_Y = jj+RY;
  //             int Im_X = ii+RX;
  //             if(Im_Y<IHEI && Im_X<IWID)
  //               {
  //                 FLOAT *PP=ac_score+Im_Y+Im_X*IHEI; //consider root rectangle size
  //                 if(sc>*PP) *PP=sc;                 //save max score
  //               }
  //           }
  //       }
  //   }
  /*************************************************************/
  /*************************************************************/
  
}



__device__
static inline int 
min_i(int x, int y) 
{return (x <= y ? x : y);}

/*************************************************/
/* atomic function dealing with double precision */
__device__ 
double 
atomicAdd_double(double *address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int  old            = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
  }while(assumed != old);
  return __longlong_as_double(old);
}
/*************************************************/

/************************************************/
/* atomic function dealing with float precision */
__device__
void
atomicAdd_float(float *address, float val)
{
  atomicAdd(address, val);      // atomicAdd must be called from "__device__" function
}
/*************************************************/

/***********************************************************/
/* function which cast from int2 to unsigned long long int */
__device__
unsigned long long int
hiloint2uint64(int hi, int lo)
{
  int combined[] = {hi, lo};
  return *reinterpret_cast<unsigned long long int*>(combined);
}
/***********************************************************/


/* declaration of texture memory */
#ifdef USE_FLOAT_AS_DECIMAL
texture<float, hipTextureType1D, hipReadModeElementType> resized_image;
#else
texture<int2, hipTextureType1D, hipReadModeElementType>  resized_image_double;
#endif
texture<int , hipTextureType1D, hipReadModeElementType>  resized_image_size;

texture<int, hipTextureType1D, hipReadModeElementType>   image_idx_incrementer;
texture<uint2, hipTextureType1D, hipReadModeElementType> hist_ptr_incrementer;


#ifndef USE_SHARED_MEM
/* no shared memory version */

extern "C"
__global__
void
calc_feature
(
 FLOAT *hist_top,
 int sbin1,
 int sbin2,
 int interval,
 int max_scale
 )
{
  /* index of each pixels */
  int x     = blockIdx.x * blockDim.x + threadIdx.x;
  int y     = blockIdx.y * blockDim.y + threadIdx.y;

  int level = blockIdx.z;
  int sbin  = (level < interval) ? sbin2 : sbin1;
  int LEN   = interval + max_scale;
 
  const FLOAT Hcos[9] = {1.0000, 0.9397, 0.7660, 0.5000, 0.1736, -0.1736, -0.5000, -0.7660, -0.9397};
  const FLOAT Hsin[9] = {0.0000, 0.3420, 0.6428, 0.8660, 0.9848, 0.9848, 0.8660, 0.6428, 0.3420};

  /* adjust pointer position */
  int                     base_index      = tex1Dfetch(image_idx_incrementer, level);
  uint2                   ptr_incrementer = tex1Dfetch(hist_ptr_incrementer, level);
  unsigned long long int  ptr_hist        = (unsigned long long int)hist_top + hiloint2uint64(ptr_incrementer.x, ptr_incrementer.y);
  FLOAT                  *hist            = (FLOAT *)ptr_hist;

  /* input size */
  const int height  = tex1Dfetch(resized_image_size, level*3);
  const int width   = tex1Dfetch(resized_image_size, level*3 + 1);
  const int dims[2] = {height, width};

  /* size of Histgrams and Norm calculation space */
  const int blocks[2] = {
    (int)floor((double)height/(double)sbin+0.5),
    (int)floor((double)width/(double)sbin+0.5)
  };
  
  /* Visible range (eliminate border blocks) */
  const int visible[2] = {blocks[0]*sbin, blocks[1]*sbin};
  

  // for (int x=1; x<visible[1]-1; x++) {
  //   for (int y=1; y<visible[0]-1; y++) {
  if (1<=x && x<visible[1]-1 && 1<=y && y<visible[0]-1 && 0<=level && level <= LEN) 
    {
      /* first color channel */
      base_index += min_i(x, dims[1]-2)*dims[0] + min_i(y, dims[0]-2);
      FLOAT dx, dy;
#ifdef USE_FLOAT_AS_DECIMAL
      {
        /* get "float" type values from texture memory */
        dy = tex1Dfetch(resized_image, base_index + 1) - tex1Dfetch(resized_image, base_index - 1) ;
        dx = tex1Dfetch(resized_image, base_index + dims[0]) - tex1Dfetch(resized_image, base_index - dims[0]) ;
      }
#else
      int2 arg1;
      int2 arg2;
      
      {
        /* get "double" type values from texture memory */
        arg1 = tex1Dfetch(resized_image_double, base_index + 1);
        arg2 = tex1Dfetch(resized_image_double, base_index - 1) ;
        dy = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
        
        arg1 = tex1Dfetch(resized_image_double, base_index + dims[0]);
        arg2 = tex1Dfetch(resized_image_double, base_index - dims[0]);
        dx = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
      }
#endif
      FLOAT  v  = dx*dx + dy*dy;
      
      /* second color channel */
      base_index += dims[0]*dims[1];
      FLOAT dx2, dy2;
#ifdef USE_FLOAT_AS_DECIMAL
      {
        /* get "float" type values from texture memory */
        dy2 = tex1Dfetch(resized_image, base_index + 1) - tex1Dfetch(resized_image, base_index - 1) ;
        dx2 = tex1Dfetch(resized_image, base_index + dims[0]) - tex1Dfetch(resized_image, base_index - dims[0]) ;
      }
#else
      {
        /* get "double" type values from texture memory */
        arg1 = tex1Dfetch(resized_image_double, base_index + 1);
        arg2 = tex1Dfetch(resized_image_double, base_index - 1) ;
        dy2 = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
        
        arg1 = tex1Dfetch(resized_image_double, base_index + dims[0]);
        arg2 = tex1Dfetch(resized_image_double, base_index - dims[0]);
        dx2 = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
      }
#endif
      FLOAT v2  = dx2*dx2 + dy2*dy2;
      
      /* third color channel */
      base_index += dims[0]*dims[1];
      FLOAT dx3, dy3;
#ifdef USE_FLOAT_AS_DECIMAL
      {
        /* get "float" type values from texture memory */
        dy3 = tex1Dfetch(resized_image, base_index + 1) - tex1Dfetch(resized_image, base_index - 1) ;
        dx3 = tex1Dfetch(resized_image, base_index + dims[0]) - tex1Dfetch(resized_image, base_index - dims[0]) ;
      }
#else
      {
        /* get "double" type values from texture memory */
        arg1 = tex1Dfetch(resized_image_double, base_index + 1);
        arg2 = tex1Dfetch(resized_image_double, base_index - 1) ;
        dy3 = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
        
        arg1 = tex1Dfetch(resized_image_double, base_index + dims[0]);
        arg2 = tex1Dfetch(resized_image_double, base_index - dims[0]);
        dx3 = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
      }
#endif
      FLOAT v3  = dx3*dx3 + dy3*dy3;
      
      /* pick channel with strongest gradient */
      if (v2 > v) {
        v  = v2;
        dx = dx2;
        dy = dy2;
      }
      if (v3 > v) {
        v  = v3;
        dx = dx3;
        dy = dy3;
      }
      
      /* snap to one of 18 orientations */
      FLOAT best_dot = 0;
      int   best_o   = 0;
      
#pragma unroll 9
      for (int o=0; o<9; o++) {
        FLOAT dot = Hcos[o]*dx + Hsin[o]*dy; 
        
        if (dot > best_dot) {
          best_dot = dot;
          best_o   = o;
        }
        else if (-dot > best_dot) {
          best_dot = -dot;
          best_o   = o + 9;
        }
      }
      
      /*add to 4 histgrams aroud pixel using linear interpolation*/
      FLOAT xp  = ((FLOAT)x+0.5)/(FLOAT)sbin - 0.5;
      FLOAT yp  = ((FLOAT)y+0.5)/(FLOAT)sbin - 0.5;
      int   ixp = (int)floor((double)xp);
      int   iyp = (int)floor((double)yp);
      FLOAT vx0 = xp - ixp;
      FLOAT vy0 = yp - iyp;
      FLOAT vx1 = 1.0 - vx0;
      FLOAT vy1 = 1.0 - vy0;
      v = sqrt((double)v);
      
#ifdef USE_FLOAT_AS_DECIMAL
      {
        if (ixp >= 0 && iyp >= 0) {
          atomicAdd_float((float *)(hist + ixp*blocks[0] + iyp + best_o*blocks[0]*blocks[1]), (float)vx1*vy1*v);
          //            *((float *)(hist + ixp*blocks[0] + iyp + best_o*blocks[0]*blocks[1])) += (float)vx1*vy1*v;
        }
        
        if (ixp+1 < blocks[1] && iyp >= 0) {
          atomicAdd_float((float *)(hist + (ixp+1)*blocks[0] + iyp + best_o*blocks[0]*blocks[1]), (float)vx0*vy1*v);
          //            *((float *)(hist + (ixp+1)*blocks[0] + iyp + best_o*blocks[0]*blocks[1])) += (float)vx0*vy1*v;
        }
        
        if (ixp >= 0 && iyp+1 < blocks[0]) {
          atomicAdd_float((float *)(hist + ixp*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1]), (float)vx1*vy0*v);
          //            *((float *)(hist + ixp*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1])) += (float)vx1*vy0*v;
        }
        
        if (ixp+1 < blocks[1] && iyp+1 < blocks[0]) {
          atomicAdd_float((float *)(hist + (ixp+1)*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1]), (float)vx0*vy0*v);
          //            *((float *)(hist + (ixp+1)*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1])) += (float)vx0*vy0*v;
        }
      }
#else
      {
        if (ixp >= 0 && iyp >= 0) {
          atomicAdd_double((double *)(hist + ixp*blocks[0] + iyp + best_o*blocks[0]*blocks[1]), (double)vx1*vy1*v);
        }
        
        if (ixp+1 < blocks[1] && iyp >= 0) {
          atomicAdd_double((double *)(hist + (ixp+1)*blocks[0] + iyp + best_o*blocks[0]*blocks[1]), (double)vx0*vy1*v);
        }
        
        if (ixp >= 0 && iyp+1 < blocks[0]) {
          atomicAdd_double((double *)(hist + ixp*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1]), (double)vx1*vy0*v);
        }
        
        if (ixp+1 < blocks[1] && iyp+1 < blocks[0]) {
          atomicAdd_double((double *)(hist + (ixp+1)*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1]), (double)vx0*vy0*v);
        }
      }
#endif
      
    }
      
  //   }
  // }
  
}

#else  // #ifndef USE_SHARED_MEM
/* use shared memory version */

extern "C"
__global__
void
calc_feature
(
 FLOAT *hist_top,
 int sbin1,
 int sbin2,
 int interval,
 int max_scale
 )
{
  int level = blockIdx.z;
  int sbin  = (level < interval) ? sbin2 : sbin1;
  int LEN   = interval + max_scale;

  /* index of each pixel */
  int x     = blockIdx.x * CELL_PER_BLOCK_Y * sbin + threadIdx.x;
  int y     = blockIdx.y * CELL_PER_BLOCK_Y * sbin + threadIdx.y;

  const FLOAT Hcos[9] = {1.0000, 0.9397, 0.7660, 0.5000, 0.1736, -0.1736, -0.5000, -0.7660, -0.9397};
  const FLOAT Hsin[9] = {0.0000, 0.3420, 0.6428, 0.8660, 0.9848, 0.9848, 0.8660, 0.6428, 0.3420};

  /* adjust pointer position */
  int                     base_index      = tex1Dfetch(image_idx_incrementer, level);
  uint2                   ptr_incrementer = tex1Dfetch(hist_ptr_incrementer, level);
  unsigned long long int  ptr_hist        = (unsigned long long int)hist_top + hiloint2uint64(ptr_incrementer.x, ptr_incrementer.y);
  FLOAT                  *hist            = (FLOAT *)ptr_hist;

  /* input resized size */
  const int height  = tex1Dfetch(resized_image_size, level*3);
  const int width   = tex1Dfetch(resized_image_size, level*3 + 1);
  const int dims[2] = {height, width};

  /* size of Histgrams and Norm calculation space */
  const int blocks[2] = {
    (int)floor((double)height/(double)sbin+0.5),
    (int)floor((double)width/(double)sbin+0.5)
  };
  
  /* Visible range (eliminate border blocks) */
  const int visible[2] = {blocks[0]*sbin, blocks[1]*sbin};

  /* shared sub histgram among threads in the same GPU block */ 
  /* 18 means orientation of histgram */
  __shared__ FLOAT sub_hist[VOTE_CELL_PER_BLOCK_X][VOTE_CELL_PER_BLOCK_Y][18]; 

  /* choose working threads in each GPU-thread-block */
  if (threadIdx.x<VOTE_CELL_PER_BLOCK_X && threadIdx.y<VOTE_CELL_PER_BLOCK_Y)
    {
#pragma unroll 18
      /* initialize sub histgram */
      for (int i=0; i<18; i++)
        sub_hist[threadIdx.x][threadIdx.y][i] = 0.f;
    }
  
  __syncthreads();
  
  
  /* calculate relative cell block index of pixel concerned */

  // for (int x=1; x<visible[1]-1; x++) {
  //   for (int y=1; y<visible[0]-1; y++) {
  if (1<=x && x<visible[1]-1 && 1<=y && y<visible[0]-1 && 0<=level && level <= LEN &&
      sbin/2<=threadIdx.x && sbin/2<=threadIdx.y &&
      threadIdx.x<VOTE_CELL_PER_BLOCK_X*sbin-sbin/2 && threadIdx.y<VOTE_CELL_PER_BLOCK_Y*sbin-sbin/2)
    {
      /* first color channel */
      base_index += min_i(x, dims[1]-2)*dims[0] + min_i(y, dims[0]-2);
      FLOAT dx, dy;
      
#ifdef USE_FLOAT_AS_DECIMAL
      {
        /* get "float" type values from texture memory */
        dy = tex1Dfetch(resized_image, base_index + 1) - tex1Dfetch(resized_image, base_index - 1) ;
        dx = tex1Dfetch(resized_image, base_index + dims[0]) - tex1Dfetch(resized_image, base_index - dims[0]) ;
      }
#else
      int2 arg1;
      int2 arg2;
      {
        /* get "double" type values from texture memory */
        arg1 = tex1Dfetch(resized_image_double, base_index + 1);
        arg2 = tex1Dfetch(resized_image_double, base_index - 1) ;
        dy = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
        
        arg1 = tex1Dfetch(resized_image_double, base_index + dims[0]);
        arg2 = tex1Dfetch(resized_image_double, base_index - dims[0]);
        dx = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
      }
#endif
      FLOAT  v  = dx*dx + dy*dy;
      
      /* second color channel */
      base_index += dims[0]*dims[1];
      FLOAT dx2, dy2;

#ifdef USE_FLOAT_AS_DECIMAL
      {
        /* get "float" type values from texture memory */
        dy2 = tex1Dfetch(resized_image, base_index + 1) - tex1Dfetch(resized_image, base_index - 1) ;
        dx2 = tex1Dfetch(resized_image, base_index + dims[0]) - tex1Dfetch(resized_image, base_index - dims[0]) ;
      }
#else
      {
        /* get "double" type values from texture memory */
        arg1 = tex1Dfetch(resized_image_double, base_index + 1);
        arg2 = tex1Dfetch(resized_image_double, base_index - 1) ;
        dy2 = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
        
        arg1 = tex1Dfetch(resized_image_double, base_index + dims[0]);
        arg2 = tex1Dfetch(resized_image_double, base_index - dims[0]);
        dx2 = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
      }
#endif
      FLOAT v2  = dx2*dx2 + dy2*dy2;
      
      /* third color channel */
      base_index += dims[0]*dims[1];
      FLOAT dx3, dy3;
#ifdef USE_FLOAT_AS_DECIMAL
      {
        /* get "float" type values from texture memory */
        dy3 = tex1Dfetch(resized_image, base_index + 1) - tex1Dfetch(resized_image, base_index - 1) ;
        dx3 = tex1Dfetch(resized_image, base_index + dims[0]) - tex1Dfetch(resized_image, base_index - dims[0]) ;
      }
#else
      {
        /* get "double" type values from texture memory */
        arg1 = tex1Dfetch(resized_image_double, base_index + 1);
        arg2 = tex1Dfetch(resized_image_double, base_index - 1) ;
        dy3 = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
        
        arg1 = tex1Dfetch(resized_image_double, base_index + dims[0]);
        arg2 = tex1Dfetch(resized_image_double, base_index - dims[0]);
        dx3 = __hiloint2double(arg1.y, arg1.x) - __hiloint2double(arg2.y, arg2.x);
      }    
#endif
      FLOAT v3  = dx3*dx3 + dy3*dy3;
      
      /* pick channel with strongest gradient */
      if (v2 > v) {
        v  = v2;
        dx = dx2;
        dy = dy2;
      }
      if (v3 > v) {
        v  = v3;
        dx = dx3;
        dy = dy3;
      }
      
      /* snap to one of 18 orientations */
      FLOAT best_dot = 0;
      int   best_o   = 0;
#pragma unroll 9
      for (int o=0; o<9; o++) {
        FLOAT dot = Hcos[o]*dx + Hsin[o]*dy; 
        
        if (dot > best_dot) {
          best_dot = dot;
          best_o   = o;
        }
        else if (-dot > best_dot) {
          best_dot = -dot;
          best_o   = o + 9;
        }
      }
      
      /*add to 4 histgrams aroud pixel using linear interpolation*/
      FLOAT xp  = ((FLOAT)x+0.5)/(FLOAT)sbin - 0.5;
      FLOAT yp  = ((FLOAT)y+0.5)/(FLOAT)sbin - 0.5;
      int   ixp = (int)floor((double)xp);
      int   iyp = (int)floor((double)yp);
      FLOAT vx0 = xp - ixp;
      FLOAT vy0 = yp - iyp;
      FLOAT vx1 = 1.0 - vx0;
      FLOAT vy1 = 1.0 - vy0;
      v = sqrt((double)v);


      /* index of histgram cell block in a GPU-thread-block */
      int sub_hist_idx_x = ixp % CELL_PER_BLOCK_X;
      int sub_hist_idx_y = iyp % CELL_PER_BLOCK_Y;
  

      
      /* vote gradient values to subhistgram */
#ifdef USE_FLOAT_AS_DECIMAL

      if (ixp >= 0 && iyp >= 0) {
        atomicAdd_float(&sub_hist[sub_hist_idx_x][sub_hist_idx_y][best_o], vx1*vy1*v);
      }

      if (ixp+1 < blocks[1] && iyp >= 0) {
        atomicAdd_float(&sub_hist[sub_hist_idx_x + 1][sub_hist_idx_y][best_o], vx0*vy1*v);
      }

      if (ixp >= 0 && iyp+1 < blocks[0]) {
        atomicAdd_float(&sub_hist[sub_hist_idx_x][sub_hist_idx_y + 1][best_o], vx1*vy0*v);
      }
      
      if (ixp+1 < blocks[1] && iyp+1 < blocks[0]) {
        atomicAdd_float(&sub_hist[sub_hist_idx_x + 1][sub_hist_idx_y + 1][best_o], vx0*vy0*v);
      }

#else

      if (ixp >= 0 && iyp >= 0) {
        atomicAdd_double(&sub_hist[sub_hist_idx_x][sub_hist_idx_y][best_o], vx1*vy1*v);
      }

      if (ixp+1 < blocks[1] && iyp >= 0) {
        atomicAdd_double(&sub_hist[sub_hist_idx_x + 1][sub_hist_idx_y][best_o], vx0*vy1*v);
      }

      if (ixp >= 0 && iyp+1 < blocks[0]) {
        atomicAdd_double(&sub_hist[sub_hist_idx_x][sub_hist_idx_y + 1][best_o], vx1*vy0*v);
      }
      
      if (ixp+1 < blocks[1] && iyp+1 < blocks[0]) {
        atomicAdd_double(&sub_hist[sub_hist_idx_x + 1][sub_hist_idx_y + 1][best_o], vx0*vy0*v);
      }

#endif      
      
    }
  
  /* synchronize threads in the same GPU block */
  __syncthreads();


  /* vote to global histgram */
  if (threadIdx.x<VOTE_CELL_PER_BLOCK_X && threadIdx.y<VOTE_CELL_PER_BLOCK_Y)
    {

      /* index of histgram cell block in all GPU-thread-blocks */
      int main_hist_idx_x = blockIdx.x * CELL_PER_BLOCK_X + threadIdx.x;
      int main_hist_idx_y = blockIdx.y * CELL_PER_BLOCK_Y + threadIdx.y;
      
      /* whether memory region to attempt to write is proper or not*/
      if (main_hist_idx_x < blocks[1]-1 && main_hist_idx_y < blocks[0]-1)
        {
#pragma unroll 18
          for (int i=0; i<18; i++)
            {
#ifdef USE_FLOAT_AS_DECIMAL

              unsigned long long int address = (unsigned long long int)hist + 
                (main_hist_idx_x*blocks[0] + main_hist_idx_y + i*blocks[0]*blocks[1])*sizeof(FLOAT);
              atomicAdd_float((FLOAT*)address, sub_hist[threadIdx.x][threadIdx.y][i]);

#else

              unsigned long long int address = (unsigned long long int)hist + 
                (main_hist_idx_x*blocks[0] + main_hist_idx_y + i*blocks[0]*blocks[1])*sizeof(FLOAT);
              atomicAdd_double((FLOAT*)address, sub_hist[threadIdx.x][threadIdx.y][i]);

#endif
            }
        }
    }
  
      //   }
      // }
      
  /*************************************************************/
  /* original source of calc_feature loop */

  // for (int x=1; x<visible[1]-1; x++) {
  //   for (int y=1; y<visible[0]-1; y++) {

  //     /* first color channel */
  //     FLOAT *s  = SRC + min_i(x, dims[1]-2)*dims[0] + min_i(y, dims[0]-2);
  //     FLOAT  dy = *(s+1) - *(s-1);
  //     FLOAT  dx = *(s+dims[0]) - *(s-dims[0]);
  //     FLOAT  v  = dx*dx + dy*dy;

  //     /* second color channel */
  //     s += dims[0]*dims[1];
  //     FLOAT dy2 = *(s+1) - *(s-1);
  //     FLOAT dx2 = *(s+dims[0]) - *(s-dims[0]);
  //     FLOAT v2  = dx2*dx2 + dy2*dy2;

  //     /* third color channel */
  //     s += dims[0]*dims[1];
  //     FLOAT dy3 = *(s+1) - *(s-1);
  //     FLOAT dx3 = *(s+dims[0]) - *(s-dims[0]);
  //     FLOAT v3  = dx3*dx3 + dy3*dy3;

  //     /* pick channel with strongest gradient */
  //     if (v2 > v) {
  //       v  = v2;
  //       dx = dx2;
  //       dy = dy2;
  //     }
  //     if (v3 > v) {
  //       v  = v3;
  //       dx = dx3;
  //       dy = dy3;
  //     }

  //     /* snap to one of 18 orientations */
  //     FLOAT best_dot = 0;
  //     int   best_o   = 0;
  //     for (int o=0; o<9; o++) {
  //       FLOAT dot = Hcos[o]*dx + Hsin[o]*dy; 

  //       if (dot > best_dot) {
  //         best_dot = dot;
  //         best_o   = o;
  //       }
  //       else if (-dot > best_dot) {
  //         best_dot = -dot;
  //         best_o   = o + 9;
  //       }

  //     }

  //     /*add to 4 histgrams aroud pixel using linear interpolation*/
  //     FLOAT xp  = ((FLOAT)x+0.5)/(FLOAT)sbin - 0.5;
  //     FLOAT yp  = ((FLOAT)y+0.5)/(FLOAT)sbin - 0.5;
  //     int   ixp = (int)floor(xp);
  //     int   iyp = (int)floor(yp);
  //     FLOAT vx0 = xp - ixp;
  //     FLOAT vy0 = yp - iyp;
  //     FLOAT vx1 = 1.0 - vx0;
  //     FLOAT vy1 = 1.0 - vy0;
  //     v = sqrt(v);

  //     if (ixp >= 0 && iyp >= 0) {
  //       *(hist + ixp*blocks[0] + iyp + best_o*blocks[0]*blocks[1]) += vx1*vy1*v;
  //     }

  //     if (ixp+1 < blocks[1] && iyp >= 0) {
  //       *(hist + (ixp+1)*blocks[0] + iyp + best_o*blocks[0]*blocks[1]) += vx0*vy1*v;
  //     }

  //     if (ixp >= 0 && iyp+1 < blocks[0]) {
  //       *(hist + ixp*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1]) += vx1*vy0*v;
  //     }

  //     if (ixp+1 < blocks[1] && iyp+1 < blocks[0]) {
  //       *(hist + (ixp+1)*blocks[0] + (iyp+1) + best_o*blocks[0]*blocks[1]) += vx0*vy0*v;
  //     }
  //   }
  // }

  /*************************************************************/
  /*************************************************************/


}

#endif  // #if 0

