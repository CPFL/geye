#include "hip/hip_runtime.h"
#include<stdio.h>
#include"for_use_GPU.h"


//thread process
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// convolve A and B(non_symmetric)
//unsigned __stdcall process(void *thread_arg) {

/********************************************/
/* function for calculating root */
/********************************************/
extern "C"
__global__
void
process_root(
 double *A,  
 double *B, 
 double *C, 
 int *A_dims_array, 
 int *B_dims_array, 
 int len,
 int interval, 
 int L_MAX,
 int *error_array,
 int error_array_num
) 
{
  int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  // int ii = threadIdx.z;
  // int level = blockIdx.z; 
  int ii = blockIdx.z % len;
  int level = blockIdx.z / len;

  
  int A_dims[3] = { A_dims_array[level*3], A_dims_array[level*3+1], A_dims_array[level*3+2] };
  int B_dims[3] = { B_dims_array[ii*3], B_dims_array[ii*3+1], B_dims_array[ii*3+2] };
  int C_dims[2] = { A_dims[0] - B_dims[0] + 1, A_dims[1] - B_dims[1] + 1 };
  
  
  if(0 <= ii && ii < len && 0 <= idx_x && idx_x < C_dims[1] && 0 <= idx_y && idx_y < C_dims[0] && interval <= level && level < L_MAX ) {    
    
    int num_features = A_dims[2];
    const int A_SQ = A_dims[0]*A_dims[1];
    const int B_SQ = B_dims[0]*B_dims[1];
    double add_val = 0;
    
    int x = idx_x;
    int y = idx_y;
    int XA0 = A_dims[0]*x;

    
    /* apply loop condition */
    for(int i=0; i<error_array_num; i++){
      if(error_array[i] == level){
        return;
      }
    }
    
    
    
    /* adjust the location of pointer of C */
    double *dst;
    unsigned long long int pointer = (unsigned long long int)C;

    for(int a=interval; a<level; a++){
      for(int b=0; b<len; b++){
        int height = A_dims_array[a*3] - B_dims_array[b*3] + 1; 
        int width = A_dims_array[a*3 + 1] - B_dims_array[b*3 + 1] + 1;
        
        /* error semantics */
        if(height < 1 || width < 1){
          printf("Invalid input in GPU\n");
          return;
        }
        
        pointer += (unsigned long long int)(height*width*sizeof(double));
       
      }
    }

    for(int b=0; b<ii; b++){
      int height = A_dims_array[level*3] - B_dims_array[b*3] + 1;
      int width  = A_dims_array[level*3 + 1] - B_dims_array[b*3 + 1] + 1;

      /* error semantics */
      if(height < 1 || width < 1){
        printf("Invalid input in GPU\n");
        return;
      }
      
      pointer += (unsigned long long int)(height*width*sizeof(double));
    }
    
    dst = (double *)pointer;
    
    
    //    if(ii==0 && idx_x==0 && idx_y==0 && level == interval){
    //       printf("C       %llu \n", (unsigned long long int)C);
    //       printf("dst     %llu \n", (unsigned long long int)dst);
    //       printf("pointer %llu \n", (unsigned long long int)pointer);
    //       // printf("sizeof  %llu \n", sizeof(unsigned long long int));
    //       printf("%llu %llu %llu %llu \n", (unsigned long long int)C, (unsigned long long int)dst, (unsigned long long int)pointer, (unsigned long long int)sizeof(unsigned long long int)); 
    //  }
    

    /* adjust the location of pointer of A */
    unsigned long long int pointerA = (unsigned long long int)A;
    for(int a=0; a<level; a++) {
      pointerA += (unsigned long long int)(A_dims_array[a*3]*A_dims_array[a*3 + 1]*A_dims_array[a*3 + 2]*sizeof(double));
    } 
    
    
    /* adjust the location of pointer of B */
    unsigned long long int pointerB = (unsigned long long int)B;
    for(int b=0; b<ii; b++) {
      pointerB += (unsigned long long int)(B_dims_array[b*3]*B_dims_array[b*3 + 1]*B_dims_array[b*3 + 2]*sizeof(double));
    } 

            
    for(int f = 0; f < num_features; f++) // num_features = 31
      {  
        //double *dst = C[ii];  
        double *A_src = (double *)pointerA + f*A_SQ;      
        double *B_src = (double *)pointerB + f*B_SQ;     
        
        //        int XA0 = 0;
        //        int x = idx_x;
        //for (int x = 0; x < C_dims[1]; x++) 
        //{		
        
        //        XA0 = A_dims[0]*x;
        double *A_src2 =A_src+XA0; 
        // XA0+=A_dims[0];
        //        int y = idx_y;
        //for (int y = 0; y < C_dims[0]; y++) 
        //{
        double val = 0;
        double *A_off = A_src2+y;
        double *B_off = B_src;
        
        for (int xp = 0; xp < B_dims[1]; xp++) 
          {
            double *A_temp = A_off;						
            double *B_temp = B_off;	  
            for (int yp = 0; yp < B_dims[0]; yp++) 	  
              {
                val += *(A_temp++) * *(B_temp++);
              }
            
            A_off+=A_dims[0];
            B_off+=B_dims[0];
            
          }			 
        
        //*(dst + (x*C_dims[0] + y)) += val;		

        add_val += val;
        // }
        //}
      }

    *(dst + (idx_x*C_dims[0] + idx_y)) += add_val;

    
    //     if(ii==0 && idx_x==0 && idx_y==0 && level == interval){
    //       printf("sizeof(double) in GPU %llu\n", (unsigned long long int)sizeof(double));
    //       printf("sizeof(double*) in GPU %llu\n", (unsigned long long int)sizeof(double*));
    //       printf("sizeof(unsigned long int) in GPU %llu\n", (unsigned long long int)sizeof(unsigned long int));
    //       printf("sizeof(unsigned long long int) in GPU %llu\n", (unsigned long long int)sizeof(unsigned long long int));
    //       printf("%f\n", *(dst + (idx_x*C_dims[0] + idx_y)));
    //       printf("%f\n", C[0]);
    //       //       printf("%llu\n", (unsigned long long int)dst + (idx_x*C_dims[0] + idx_y));
    //       //       printf("%llu\n\n", (unsigned long long int)C);
    
    //     } 
  
  }

    
  return;
}



/********************************************/
/* function for calculating part */
/********************************************/
extern "C"
__global__
void
process_part(
 double *A,  
 double *B, 
 double *C, 
 int *A_dims_array, 
 int *B_dims_array, 
 int len,
 int interval, 
 int L_MAX,
 int *error_array,
 int error_array_num
) 
{


  int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  // int ii = threadIdx.z;
  // int level = blockIdx.z; 
  int ii = blockIdx.z % len;
  int level = blockIdx.z / len; 

  int A_dims[3] = { A_dims_array[level*3], A_dims_array[level*3+1], A_dims_array[level*3+2] };
  int B_dims[3] = { B_dims_array[ii*3], B_dims_array[ii*3+1], B_dims_array[ii*3+2] };
  int C_dims[2] = { A_dims[0] - B_dims[0] + 1, A_dims[1] - B_dims[1] + 1 };


  if(0 <= ii && ii < len && 0 <= idx_x && idx_x < C_dims[1] && 0 <= idx_y && idx_y < C_dims[0] && 0 <= level && level < (L_MAX - interval) ) {

    int num_features = A_dims[2];
    const int A_SQ = A_dims[0]*A_dims[1];
    const int B_SQ = B_dims[0]*B_dims[1];
    double add_val = 0;

    int x = idx_x;
    int y = idx_y;
    int XA0 = A_dims[0]*x;
    
    /* apply loop condition */
    for(int i=0; i<error_array_num; i++){
      if(error_array[i] == level)
        return;
    }
    

    /* adjust the location of pointer of C */
    double *dst;
    unsigned long long int pointer = (unsigned long long int)C;
    for(int a=0; a<level; a++) {
      //for(int b=0; b<ii; b++){
      for(int b=0; b<len; b++){
        int height = A_dims_array[a*3] - B_dims_array[b*3] + 1;
        int width = A_dims_array[a*3 + 1] - B_dims_array[b*3 + 1] + 1;
        
        /* error semantics */
        if(height < 1 || width < 1){
          printf("Invalid input in GPU\n");
          return;
        }
        
        pointer += (unsigned long long int)(height*width*sizeof(double));
      }
    }

    for(int b=0; b<ii; b++){
      int height = A_dims_array[level*3] - B_dims_array[b*3] + 1;
      int width  = A_dims_array[level*3 + 1] - B_dims_array[b*3 + 1] + 1;

       /* error semantics */
        if(height < 1 || width < 1){
          printf("Invalid input in GPU\n");
          return;
        }

      pointer += (unsigned long long int)(height*width*sizeof(double));
    }
    

    dst = (double *)pointer;

    /* adjust the location of pointer of A */
    unsigned long long int pointerA = (unsigned long long int)A;
    for(int a=0; a<level; a++) {
      pointerA += (unsigned long long int)(A_dims_array[a*3]*A_dims_array[a*3 + 1]*A_dims_array[a*3 + 2]*sizeof(double));
    } 
    
    /* adjust the location of pointer of B */
    unsigned long long int pointerB = (unsigned long long int)B;
    for(int b=0; b<ii; b++) {
      pointerB += (unsigned long long int)(B_dims_array[b*3]*B_dims_array[b*3 + 1]*B_dims_array[b*3 + 2]*sizeof(double));
    } 
    
    for(int f = 0; f < num_features; f++) // num_features = 31
      {  
        //double *dst = C[ii];  
        double *A_src = (double *)pointerA + f*A_SQ;      
        double *B_src = (double *)pointerB + f*B_SQ;     
        
        //        int XA0 = 0;
        //        int x = idx_x;
        //for (int x = 0; x < C_dims[1]; x++) 
        //{		
        
        //        XA0 = A_dims[0]*x;
        double *A_src2 =A_src+XA0; 
        // XA0+=A_dims[0];
        //        int y = idx_y;
        //for (int y = 0; y < C_dims[0]; y++) 
        //{
        double val = 0;
        double *A_off = A_src2+y;
        double *B_off = B_src;
        
        for (int xp = 0; xp < B_dims[1]; xp++) 
          {
            double *A_temp = A_off;						
            double *B_temp = B_off;	  
            for (int yp = 0; yp < B_dims[0]; yp++) 	  
              {
                val += *(A_temp++) * *(B_temp++);
              }
            
            A_off+=A_dims[0];
            B_off+=B_dims[0];
            
          }			 
        
        //*(dst + (x*C_dims[0] + y)) += val;		
        add_val += val;
        // }
        //}
      }


    *(dst + (idx_x*C_dims[0] + idx_y)) += add_val;

  }
  
  return;
}
